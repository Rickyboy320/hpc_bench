#include "hip/hip_runtime.h"
#include <pthread.h>

#include "common.h"

#define cudaCheck(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"CUDA: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}


__global__ void
vectorAdd(const float *A, float *C, int size, int offset)
{
    // TODO: offset properly.
    int i = blockDim.x * blockIdx.x + threadIdx.x + offset;
    int prev = i == 0 ? N - 1 : i - 1;
    int next = i == N - 1 ? 0 : i + 1;

    if (i < offset + size)
    {
        C[i] = A[prev] + A[i] + A[next];
    }
}

// Allocate cuda memory and pin host memory (required for async stream).
void alloc_cuda(task_t* task)
{
    hipSetDevice(task->cuda.id);

    task->cuda.size = N * sizeof(float);

    // Allocate the device vectors
    cudaCheck(hipMalloc((void **)&task->cuda.A, task->cuda.size));
    cudaCheck(hipMalloc((void **)&task->cuda.C, task->cuda.size));
}

// Deallocate cuda memory and unpin host memory.
void dealloc_cuda(task_t* task)
{
    hipSetDevice(task->cuda.id);

    // Free device global memory
    cudaCheck(hipFree(task->cuda.A));
    cudaCheck(hipFree(task->cuda.C));
}

// Run the cuda task (on the 'thread stream').
void* run_cuda(void* v_task)
{
    task_t* task = (task_t*) v_task;

    hipSetDevice(task->cuda.id);

    while(!task->done) {
        // Copy the host input vectors A and B H2D.
        cudaCheck(hipMemcpy(task->cuda.A, task->A, task->cuda.size, hipMemcpyHostToDevice));

        // Launch the vector-add CUDA Kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (task->size + threadsPerBlock - 1) / threadsPerBlock;

        vectorAdd<<<blocksPerGrid, threadsPerBlock, 0>>>(task->cuda.A, task->cuda.C, task->size, task->offset);

        // Copy the device result vector D2H.
        cudaCheck(hipMemcpy(task->C, task->cuda.C, task->cuda.size, hipMemcpyDeviceToHost));

        cudaCheck(hipDeviceSynchronize());

        task->barrier->wait();
    }

    pthread_exit(NULL);
}

// Get the number of available GPUs.
int init_cuda()
{
    int gpu_count;
    hipError_t cerr = hipGetDeviceCount(&gpu_count);
    if(cerr == hipErrorNoDevice) {
        gpu_count = 0;
    } else {
        cudaCheck(cerr);
    }

    return gpu_count;
}