#include "hip/hip_runtime.h"
#include <pthread.h>

#include "common.h"

#define cudaCheck(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"CUDA: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}


__global__ void
vectorAdd(const float *A, float *C, int size, int offset, int inset)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < size)
    {
        C[i] = A[i + inset];
        if(i + offset > 0) { C[i] += A[i - 1 + inset]; }
        if(i + offset < N) { C[i] += A[i + 1 + inset]; }
    }
}

// Allocate cuda memory and pin host memory (required for async stream).
void alloc_cuda(task_t* task)
{
    hipSetDevice(task->cuda.id);

    task->cuda.size = task->size * sizeof(float);

    // Allocate the device vectors
    cudaCheck(hipMalloc((void **)&task->cuda.A, task->cuda.size + 2 * sizeof(float))); // Plus 'imported' neighbours.
    cudaCheck(hipMalloc((void **)&task->cuda.C, task->cuda.size));
}

// Deallocate cuda memory and unpin host memory.
void dealloc_cuda(task_t* task)
{
    hipSetDevice(task->cuda.id);

    // Free device global memory
    cudaCheck(hipFree(task->cuda.A));
    cudaCheck(hipFree(task->cuda.C));
}

// Run the cuda task (on the 'thread stream').
void* run_cuda(void* v_task)
{
    task_t* task = (task_t*) v_task;

    printf("Setting device: %d\n", task->cuda.id);
    hipSetDevice(task->cuda.id);

    while(!task->done) {
        printf("Cuda memcpy h2d: %d\n", task->cuda.id);
        // Copy the host input vectors A and B H2D.

        int inset = 0;
        if(task->offset == 0) {
            cudaCheck(hipMemcpy(task->cuda.A, task->A, task->cuda.size + sizeof(float), hipMemcpyHostToDevice));
        } else if(task->offset + task->size >= N) {
            cudaCheck(hipMemcpy(task->cuda.A, &task->A[-1], task->cuda.size + sizeof(float), hipMemcpyHostToDevice));
            inset = 1;
        } else {
            cudaCheck(hipMemcpy(task->cuda.A, &task->A[-1], task->cuda.size + 2 * sizeof(float), hipMemcpyHostToDevice));
            inset = 1;
        }

        // Launch the vector-add CUDA Kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (task->size + threadsPerBlock - 1) / threadsPerBlock;

        printf("Cuda kernel: %d\n", task->cuda.id);
        vectorAdd<<<blocksPerGrid, threadsPerBlock, 0>>>(task->cuda.A, task->cuda.C, task->size, task->offset, inset);

        // Copy the device result vector D2H.
        printf("Cuda memcpy d2h: %d\n", task->cuda.id);
        cudaCheck(hipMemcpy(task->C, task->cuda.C, task->cuda.size, hipMemcpyDeviceToHost));

        printf("Cuda sync: %d\n", task->cuda.id);
        cudaCheck(hipDeviceSynchronize());

        printf("Waiting barrier Cuda: %d\n", task->cuda.id);
        task->barrier->wait();
        task->barrier->wait();
    }

    printf("cud done: %d\n", task->cuda.id);
    pthread_exit(NULL);
}

// Get the number of available GPUs.
int init_cuda()
{
    int gpu_count;
    hipError_t cerr = hipGetDeviceCount(&gpu_count);
    if(cerr == hipErrorNoDevice) {
        gpu_count = 0;
    } else {
        cudaCheck(cerr);
    }

    return gpu_count;
}