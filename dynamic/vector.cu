#include "hip/hip_runtime.h"
#include <pthread.h>

#include "manager.h"
#include "common.h"

#define cudaCheck(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"CUDA: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}


__global__ void
vectorAdd(const float *A, float *C, int size, int inset)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < size)
    {
        C[i] = A[i + inset] + A[i - 1 + inset] + A[i + 1 + inset];
    }
}

// Allocate cuda memory and pin host memory (required for async stream).
void alloc_cuda(task_t* task)
{
    hipSetDevice(task->cuda.id);

    // Allocate the device vectors
    cudaCheck(hipMalloc((void **)&task->cuda.A, (task->size + 2) * sizeof(float))); // Plus 'imported' neighbours.
    cudaCheck(hipMalloc((void **)&task->cuda.C, task->size * sizeof(float)));
}

// Deallocate cuda memory and unpin host memory.
void dealloc_cuda(task_t* task)
{
    hipSetDevice(task->cuda.id);

    // Free device global memory
    cudaCheck(hipFree(task->cuda.A));
    cudaCheck(hipFree(task->cuda.C));
}

// Run the cuda task (on the 'thread stream').
void* run_cuda(void* v_task)
{
    task_t* task = (task_t*) v_task;
    int iteration = task->start_iteration;
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    printf("Setting device: %d\n", task->cuda.id);
    hipSetDevice(task->cuda.id);

    for(; iteration < CYCLES; iteration++) {
        // Copy the host input vectors A and B H2D.

        printf("A: %p, cudaA: %p, size: %d\n", task->cuda.A, &task->A[-1], (task->size + 2) * sizeof(float));

        int inset = 0;
        cudaCheck(hipMemcpy(task->cuda.A, &task->A[-1], (task->size + 2) * sizeof(float), hipMemcpyHostToDevice));
        inset = 1;

        // Launch the vector-add CUDA Kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (task->size + threadsPerBlock - 1) / threadsPerBlock;

        vectorAdd<<<blocksPerGrid, threadsPerBlock, 0>>>(task->cuda.A, task->cuda.C, task->size, inset);

        // Copy the device result vector D2H.
        cudaCheck(hipMemcpy(task->C, task->cuda.C, task->size * sizeof(float), hipMemcpyDeviceToHost));

        cudaCheck(hipDeviceSynchronize());

        printf("cuda wait\n");
        task->barrier->wait();

        // Switch buffers
        for(int j = 0; j < task->size; j++) {
            printf("C%d: (%d) [%d] %d: %f\n", iteration, rank, task->id, j, task->C[j]);

            task->A[j] = task->C[j];
        }

        printf("(%d) Updating neighbours\n", rank);
        std::vector<MPI_Receive_req> requests;
        std::vector<int> types;
        fetch_and_update_neighbours(rank, task, requests, types, false);

        // Split
        // if(will_split) {
        //      // Arbitrarily (as a test) decide to split.
        //      split(task, rank, target);
        // }

        MPI_Status statuses[requests.size()];
        if(!requests.empty()) {
            MPI_Recv_all(requests, MPI_COMM_WORLD, statuses);
        }

        for(int i = 0; i < requests.size(); i++) {
            if(statuses[i].MPI_TAG == SPLIT) {
                // Received notification of split of target. Will update refs.
                if(types[i] == NEXT_TYPE) {
                    int start = task->offset + task->size;
                    MPI_Send(&start, 1, MPI_INT, 0, LOOKUP, MPI_COMM_WORLD);
                    int new_rank;
                    MPI_Recv(&new_rank, 1, MPI_INT, 0, LOOKUP, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                    task->next.rank = new_rank;
                } else if(types[i] == PREV_TYPE) {
                    int start = task->offset - 1;
                    MPI_Send(&start, 1, MPI_INT, 0, LOOKUP, MPI_COMM_WORLD);
                    int new_rank;
                    MPI_Recv(&new_rank, 1, MPI_INT, 0, LOOKUP, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                    task->prev.rank = new_rank;
                } else {
                    throw std::runtime_error("CUDA: Invalid SPLIT type received.");
                }
            }
        }

        for(int j = -1; j < task->size + 1; j++) {
            printf("A @ C%d: (%d) [%d] %d: %f\n", iteration, rank, task->id, j, task->A[j]);
        }

        task->barrier->wait();
        //MPI Barrier @ mainthread
        task->barrier->wait();
    }

    printf("cuda done\n");
    pthread_exit(NULL);
}

// Get the number of available GPUs.
int init_cuda()
{
    int gpu_count;
    hipError_t cerr = hipGetDeviceCount(&gpu_count);
    if(cerr == hipErrorNoDevice) {
        gpu_count = 0;
    } else {
        cudaCheck(cerr);
    }

    return gpu_count;
}