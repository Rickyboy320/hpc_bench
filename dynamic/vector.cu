#include "hip/hip_runtime.h"
#include <pthread.h>

#include "common.h"

#define cudaCheck(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"CUDA: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}


__global__ void
vectorAdd(const float *A, float *C, int size, int inset)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < size)
    {
        C[i] = A[i + inset] + A[i - 1 + inset] + A[i + 1 + inset];
    }
}

// Allocate cuda memory and pin host memory (required for async stream).
void alloc_cuda(task_t* task)
{
    hipSetDevice(task->cuda.id);

    task->cuda.size = task->size * sizeof(float);

    // Allocate the device vectors
    cudaCheck(hipMalloc((void **)&task->cuda.A, task->cuda.size + 2 * sizeof(float))); // Plus 'imported' neighbours.
    cudaCheck(hipMalloc((void **)&task->cuda.C, task->cuda.size));
}

// Deallocate cuda memory and unpin host memory.
void dealloc_cuda(task_t* task)
{
    hipSetDevice(task->cuda.id);

    // Free device global memory
    cudaCheck(hipFree(task->cuda.A));
    cudaCheck(hipFree(task->cuda.C));
}

// Run the cuda task (on the 'thread stream').
void* run_cuda(void* v_task)
{
    task_t* task = (task_t*) v_task;
    int iteration = task->start_iteration;
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    printf("Setting device: %d\n", task->cuda.id);
    hipSetDevice(task->cuda.id);

    for(; iteration < CYCLES; iteration++) {
        // Copy the host input vectors A and B H2D.

        printf("A: %p, cudaA: %p, size: %d\n", task->cuda.A, &task->A[-1], task->cuda.size + 2*sizeof(float));

        int inset = 0;
        cudaCheck(hipMemcpy(task->cuda.A, &task->A[-1], task->cuda.size + 2 * sizeof(float), hipMemcpyHostToDevice));
        inset = 1;

        // Launch the vector-add CUDA Kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (task->size + threadsPerBlock - 1) / threadsPerBlock;

        vectorAdd<<<blocksPerGrid, threadsPerBlock, 0>>>(task->cuda.A, task->cuda.C, task->size, inset);

        // Copy the device result vector D2H.
        cudaCheck(hipMemcpy(task->C, task->cuda.C, task->cuda.size, hipMemcpyDeviceToHost));

        cudaCheck(hipDeviceSynchronize());

        printf("cuda wait\n");
        task->barrier->wait();

        // Switch buffers
        for(int j = 0; j < task->size; j++) {
            printf("C%d: (%d) %d: %f\n", iteration, rank, j, task->C[j]);

            task->A[j] = task->C[j];
        }

        printf("(%d) Updating neighbours\n", rank);
        std::vector<MPI_Request> requests;
        fetch_and_update_neighbours(rank, task, requests);
        // TODO: now this deadlocks because 3rd patch does not know that the patch changed.

        MPI_Status* statuses;
        MPI_Waitall(requests.size(), &requests[0], statuses);


        task->barrier->wait();
        //MPI Barrier @ mainthread
        task->barrier->wait();
    }

    printf("cuda done\n");
    pthread_exit(NULL);
}

// Get the number of available GPUs.
int init_cuda()
{
    int gpu_count;
    hipError_t cerr = hipGetDeviceCount(&gpu_count);
    if(cerr == hipErrorNoDevice) {
        gpu_count = 0;
    } else {
        cudaCheck(cerr);
    }

    return gpu_count;
}