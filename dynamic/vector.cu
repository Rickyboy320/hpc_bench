#include "hip/hip_runtime.h"
#include <pthread.h>

#include "common.h"

#define cudaCheck(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"CUDA: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}


__global__ void
vectorAdd(const float *A, float *C, int size, int inset)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < size)
    {
        C[i] = A[i + inset] + A[i - 1 + inset] + A[i + 1 + inset];
    }
}

// Allocate cuda memory and pin host memory (required for async stream).
void alloc_cuda(task_t* task)
{
    hipSetDevice(task->cuda.id);

    task->cuda.size = task->size * sizeof(float);

    // Allocate the device vectors
    cudaCheck(hipMalloc((void **)&task->cuda.A, task->cuda.size + 2 * sizeof(float))); // Plus 'imported' neighbours.
    cudaCheck(hipMalloc((void **)&task->cuda.C, task->cuda.size));
}

// Deallocate cuda memory and unpin host memory.
void dealloc_cuda(task_t* task)
{
    hipSetDevice(task->cuda.id);

    // Free device global memory
    cudaCheck(hipFree(task->cuda.A));
    cudaCheck(hipFree(task->cuda.C));
}

// Run the cuda task (on the 'thread stream').
void* run_cuda(void* v_task)
{
    task_t* task = (task_t*) v_task;

    printf("Setting device: %d\n", task->cuda.id);
    hipSetDevice(task->cuda.id);

    while(!task->done) {
        // Copy the host input vectors A and B H2D.

        printf("A: %p, cudaA: %p, size: %d\n", task->cuda.A, &task->A[-1], task->cuda.size + 2*sizeof(float));

        int inset = 0;
        cudaCheck(hipMemcpy(task->cuda.A, &task->A[-1], task->cuda.size + 2 * sizeof(float), hipMemcpyHostToDevice));
        inset = 1;

        // Launch the vector-add CUDA Kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (task->size + threadsPerBlock - 1) / threadsPerBlock;

        vectorAdd<<<blocksPerGrid, threadsPerBlock, 0>>>(task->cuda.A, task->cuda.C, task->size, inset);

        // Copy the device result vector D2H.
        cudaCheck(hipMemcpy(task->C, task->cuda.C, task->cuda.size, hipMemcpyDeviceToHost));

        cudaCheck(hipDeviceSynchronize());

        printf("cuda wait\n");
        task->barrier->wait();
        task->barrier->wait();
    }

    printf("cuda done\n");
    pthread_exit(NULL);
}

// Get the number of available GPUs.
int init_cuda()
{
    int gpu_count;
    hipError_t cerr = hipGetDeviceCount(&gpu_count);
    if(cerr == hipErrorNoDevice) {
        gpu_count = 0;
    } else {
        cudaCheck(cerr);
    }

    return gpu_count;
}