#include "hip/hip_runtime.h"
#include <pthread.h>

#include "common.h"

#define cudaCheck(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"CUDA: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}


__global__ void
vectorAdd(const float *A, const float *B, float *C, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size)
    {
        C[i] = A[i] + B[i];
    }
}

void run_cuda_stream(task_t task, hipStream_t stream)
{
    int size = task.size;
    size_t byteSize = size * sizeof(float);

    // Allocate the device vectors
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;
    cudaCheck(hipHostRegister(task.A, byteSize, 0));
    cudaCheck(hipHostRegister(task.B, byteSize, 0));
    cudaCheck(hipHostRegister(task.C, byteSize, 0));

    cudaCheck(hipMalloc((void **)&d_A, byteSize));
    cudaCheck(hipMalloc((void **)&d_B, byteSize));
    cudaCheck(hipMalloc((void **)&d_C, byteSize));

    // Copy the host input vectors A and B H2D.
    cudaCheck(hipMemcpyAsync(d_A, task.A, byteSize, hipMemcpyHostToDevice, stream));
    cudaCheck(hipMemcpyAsync(d_B, task.B, byteSize, hipMemcpyHostToDevice, stream));

    // Launch the vector-add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, size);

    cudaCheck(hipStreamSynchronize(stream));

    // Copy the device result vector D2H.
    cudaCheck(hipMemcpyAsync(task.C, d_C, byteSize, hipMemcpyDeviceToHost, stream));

    // Free device global memory
    cudaCheck(hipHostUnregister(task.A));
    cudaCheck(hipHostUnregister(task.B));
    cudaCheck(hipHostUnregister(task.C));

    cudaCheck(hipFree(d_A));
    cudaCheck(hipFree(d_B));
    cudaCheck(hipFree(d_C));

}

void* run_cuda(void* v_task)
{
    task_t* task = (task_t*) v_task;

    run_cuda_stream(*task, hipStreamPerThread);

    if(task->is_threads)
    {
        pthread_exit(NULL);
    }
    else
    {
        return NULL;
    }
}

hipStream_t* run_cuda_streams(int gpu_count, task_t tasks[])
{
    hipStream_t* streams = (hipStream_t*) malloc(sizeof(hipStream_t) * gpu_count);
    for (int i = 0; i < gpu_count; i++)
    {
        cudaCheck(hipStreamCreate(&streams[i]));

        run_cuda_stream(tasks[i + 1], streams[i]);
    }

    return streams;
}

void sync_cuda_streams(int gpu_count, hipStream_t* streams)
{
    for(int i = 0; i < gpu_count; i++)
    {
        cudaCheck(hipStreamDestroy(streams[i]));
    }

    free(streams);
}

int init_cuda()
{
    int gpu_count;
    hipError_t cerr = hipGetDeviceCount(&gpu_count);
    if(cerr == hipErrorNoDevice) {
        gpu_count = 0;
    } else {
        cudaCheck(cerr);
    }

    return gpu_count;
}