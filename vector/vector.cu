#include "hip/hip_runtime.h"
#include <pthread.h>

#include "common.h"

#define cudaCheck(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"CUDA: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}


__global__ void
vectorAdd(const float *A, const float *B, float *C, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size)
    {
        C[i] = A[i] + B[i];
    }
}

// Allocate cuda memory and pin host memory (required for async stream).
void alloc_cuda(task_t* task)
{
    int size = task->size;

    task->cudamem.size = size * sizeof(float);

    // Allocate the device vectors
    cudaCheck(hipHostRegister(task->A, task->cudamem.size, 0));
    cudaCheck(hipHostRegister(task->B, task->cudamem.size, 0));
    cudaCheck(hipHostRegister(task->C, task->cudamem.size, 0));

    cudaCheck(hipMalloc((void **)&task->cudamem.A, task->cudamem.size));
    cudaCheck(hipMalloc((void **)&task->cudamem.B, task->cudamem.size));
    cudaCheck(hipMalloc((void **)&task->cudamem.C, task->cudamem.size));
}

// Deallocate cuda memory and unpin host memory.
void dealloc_cuda(task_t* task)
{
    // Free device global memory
    cudaCheck(hipHostUnregister(task->A));
    cudaCheck(hipHostUnregister(task->B));
    cudaCheck(hipHostUnregister(task->C));

    cudaCheck(hipFree(task->cudamem.A));
    cudaCheck(hipFree(task->cudamem.B));
    cudaCheck(hipFree(task->cudamem.C));
}

// Run cuda kernel asynchronously on the given stream.
void run_cuda_stream(task_t task, hipStream_t stream)
{
    // Copy the host input vectors A and B H2D.
    cudaCheck(hipMemcpyAsync(task.cudamem.A, task.A, task.cudamem.size, hipMemcpyHostToDevice, stream));
    cudaCheck(hipMemcpyAsync(task.cudamem.B, task.B, task.cudamem.size, hipMemcpyHostToDevice, stream));

    // Launch the vector-add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (task.size + threadsPerBlock - 1) / threadsPerBlock;

    vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(task.cudamem.A, task.cudamem.B, task.cudamem.C, task.size);

    // Copy the device result vector D2H.
    cudaCheck(hipMemcpyAsync(task.C, task.cudamem.C, task.cudamem.size, hipMemcpyDeviceToHost, stream));
}

// Run the cuda task (on the 'thread stream').
void* run_cuda(void* v_task)
{
    task_t* task = (task_t*) v_task;

    run_cuda_stream(*task, hipStreamPerThread);
    cudaCheck(hipStreamSynchronize(hipStreamPerThread));

    if(task->is_threads)
    {
        pthread_exit(NULL);
    }
    else
    {
        return NULL;
    }
}

// Create and run streams for each of the tasks.
hipStream_t* run_cuda_streams(int gpu_count, task_t tasks[])
{
    hipStream_t* streams = (hipStream_t*) malloc(sizeof(hipStream_t) * gpu_count);
    for (int i = 0; i < gpu_count; i++)
    {
        cudaCheck(hipStreamCreate(&streams[i]));

        run_cuda_stream(tasks[i + 1], streams[i]);
    }

    return streams;
}

// Syncrhonize and delete all streams.
void sync_cuda_streams(int gpu_count, hipStream_t* streams)
{
    for(int i = 0; i < gpu_count; i++)
    {
        cudaCheck(hipStreamSynchronize(streams[i]));
        cudaCheck(hipStreamDestroy(streams[i]));
    }

    free(streams);
}

// Get the number of available GPUs.
int init_cuda()
{
    int gpu_count;
    hipError_t cerr = hipGetDeviceCount(&gpu_count);
    if(cerr == hipErrorNoDevice) {
        gpu_count = 0;
    } else {
        cudaCheck(cerr);
    }

    return gpu_count;
}